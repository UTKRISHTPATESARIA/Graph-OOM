_
#include "subgraph.cuh"
#include "gpu_error_check.cuh"
#include "graph.cuh"
//#include <hip/hip_runtime_api.h>

typedef unsigned long long ull;

template <class E>
Subgraph<E>::Subgraph(uint num_nodes, uint num_edges)
{
	/*hipProfilerStart();
	hipError_t error;
	hipDeviceProp_t dev;
	int deviceID;
	int oversub_percentage = 70;
	ull available_memory;
	hipGetDevice(&deviceID);
	error = hipGetDeviceProperties(&dev, deviceID);
	if(error != hipSuccess)
	{
		printf("Error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	hipProfilerStop();
	
	*/
	//max_partition_size = 196902099;
	/*hipProfilerStart();
	hipError_t error;
	hipDeviceProp_t dev;
	int deviceID;
	hipGetDevice(&deviceID);
	error = hipGetDeviceProperties(&dev, deviceID);
	if(error != hipSuccess)
	{
		printf("Error: %s\n", hipGetErrorString(error));
		exit(-1);
	}
	hipProfilerStop();*/
	
	//max_partition_size = 0.5 * (dev.totalGlobalMem - 8*4*num_nodes) / sizeof(E);
	int oversub_percentage = 60;
	ull available_memory;
	available_memory = 25445793792 - ((25445793792/100)*oversub_percentage);
	max_partition_size = 0.9 * (available_memory - 8*4*num_nodes) / sizeof(E);

	//available_memory = 0.5*
	//max_partition_size = 0.5 * (- 8*4*num_nodes) / sizeof(E);
	
	if(max_partition_size > DIST_INFINITY)
		max_partition_size = DIST_INFINITY;
	
	//cout << "Max Partition Size: " << max_partition_size << endl;
	
	this->num_nodes = num_nodes;
	this->num_edges = num_edges;

	
	gpuErrorcheck(hipHostMalloc(&activeNodes, num_nodes * sizeof(uint)));
	gpuErrorcheck(hipHostMalloc(&activeNodesPointer, (num_nodes+1) * sizeof(uint)));
	gpuErrorcheck(hipHostMalloc(&activeEdgeList, num_edges * sizeof(E)));

	gpuErrorcheck(hipMalloc(&d_activeNodes, num_nodes * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_activeNodesPointer, (num_nodes+1) * sizeof(unsigned int)));
	gpuErrorcheck(hipMalloc(&d_activeEdgeList, (max_partition_size) * sizeof(E)));
}


template class Subgraph<OutEdge>;
template class Subgraph<OutEdgeWeighted>;

// For initialization with one active node
//unsigned int numActiveNodes = 1;
//subgraph.activeNodes[0] = SOURCE_NODE;
//for(unsigned int i=graph.nodePointer[SOURCE_NODE], j=0; i<graph.nodePointer[SOURCE_NODE] + graph.outDegree[SOURCE_NODE]; i++, j++)
//	subgraph.activeEdgeList[j] = graph.edgeList[i];
//subgraph.activeNodesPointer[0] = 0;
//subgraph.activeNodesPointer[1] = graph.outDegree[SOURCE_NODE];
//gpuErrorcheck(hipMemcpy(subgraph.d_activeNodes, subgraph.activeNodes, numActiveNodes * sizeof(unsigned int), hipMemcpyHostToDevice));
//gpuErrorcheck(hipMemcpy(subgraph.d_activeNodesPointer, subgraph.activeNodesPointer, (numActiveNodes+1) * sizeof(unsigned int), hipMemcpyHostToDevice));


